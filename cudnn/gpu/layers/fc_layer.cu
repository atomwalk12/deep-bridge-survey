#include "hip/hip_runtime.h"
#include "fc_layer.h"
#include "../core/utils.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <hipDNN.h>
#include <random>

FCLayer::FCLayer(hipdnnHandle_t &cudnn_handle,
                 int batch_size,
                 int input_features,
                 int output_features)
    : Layer(cudnn_handle),
      batch_size(batch_size),
      input_features(input_features),
      output_features(output_features)
{

    hipblasStatus_t status = hipblasCreate(&cublas_handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("cuBLAS initialization failed: %d\n", status);
        exit(1);
    }
    createDescriptors();

    // Allocate and initialize weights
    size_t weight_size = input_features * output_features * sizeof(float);
    hipMallocManaged(&weights, weight_size);
    hipMallocManaged(&weight_gradients, weight_size);

    std::random_device rd;
    std::mt19937 gen(rd());

    // He initialization - using input_features as fan_in
    float std = sqrt(2.0f / input_features);
    std::normal_distribution<float> distribution(0.0f, std);

    for (int i = 0; i < input_features * output_features; i++)
    {
        weights[i] = distribution(gen);
    }

    zeroGradients();
}

void FCLayer::createDescriptors()
{
    hipdnnCreateTensorDescriptor(&input_descriptor);
    hipdnnCreateTensorDescriptor(&output_descriptor);

    hipdnnSetTensor4dDescriptor(
        input_descriptor,
        HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT,
        batch_size, input_features, 1, 1);

    hipdnnSetTensor4dDescriptor(
        output_descriptor,
        HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT,
        batch_size, output_features, 1, 1);
}

void FCLayer::forward(float *input, float *output)
{
    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Perform matrix multiplication: output = weights * input
    // Goal: output = weights × input
    hipblasStatus_t status = hipblasSgemm(cublas_handle,
                                        HIPBLAS_OP_N,     // weights as-is: [output_features × input_features]
                                        HIPBLAS_OP_N,     // input as-is: [input_features × batch_size]
                                        output_features, // rows of A
                                        batch_size,      // cols of A
                                        input_features,  // cols of A and rows of B
                                        &alpha,
                                        weights, // [output_features × input_features]
                                        output_features,
                                        input, // [input_features × batch_size]
                                        input_features,
                                        &beta,
                                        output, // [output_features × batch_size]
                                        output_features);

    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        printf("CUBLAS error: %s\n", cublasGetStatusString(status));
        exit(1);
    }

    // Debug output values
    float *host_output = new float[output_features * batch_size];
    hipMemcpy(host_output, output, output_features * batch_size * sizeof(float), hipMemcpyDeviceToHost);
}

void FCLayer::backwardInput(float *input_gradient, float *output_gradient)
{
    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Perform matrix multiplication: input_gradient = weights^T × output_gradient
    hipblasStatus_t status = hipblasSgemm(cublas_handle,
                                        HIPBLAS_OP_T,     // Need weights transposed
                                        HIPBLAS_OP_N,     // output_gradient as-is
                                        input_features,  // rows of result
                                        batch_size,      // cols of result
                                        output_features, // inner dimension
                                        &alpha,
                                        weights, // [output_features × input_features]
                                        output_features,
                                        output_gradient, // [output_features × batch_size]
                                        output_features,
                                        &beta,
                                        input_gradient, // [input_features × batch_size]
                                        input_features);

    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        printf("CUBLAS error: %s\n", cublasGetStatusString(status));
        exit(1);
    }

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(err));
        exit(1);
    }
}

void FCLayer::backwardParams(float *input, float *output_gradient)
{
    const float alpha = 1.0f;
    const float beta = 1.0f; // Accumulate gradients

    // Perform matrix multiplication: weight_gradients = output_gradient × input^T
    // See https://docs.nvidia.com/cuda/cublas/ -> hipblasSgemm
    hipblasStatus_t status = hipblasSgemm(cublas_handle,
                                        HIPBLAS_OP_N,     // output_gradient as-is
                                        HIPBLAS_OP_T,     // Need input transposed
                                        output_features, // rows of result
                                        input_features,  // cols of result
                                        batch_size,      // inner dimension
                                        &alpha,
                                        output_gradient, // [output_features × batch_size]
                                        output_features,
                                        input, // [input_features × batch_size]
                                        input_features,
                                        &beta,
                                        weight_gradients, // [output_features × input_features]
                                        output_features);

    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        printf("CUBLAS error: %s\n", cublasGetStatusString(status));
        exit(1);
    }

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(err));
        exit(1);
    }
}

void FCLayer::zeroGradients()
{
    hipMemset(weight_gradients, 0, getWeightSize() * sizeof(float));
}

void FCLayer::destroyDescriptors()
{
    hipdnnDestroyTensorDescriptor(input_descriptor);
    hipdnnDestroyTensorDescriptor(output_descriptor);
    hipFree(weights);
    hipFree(weight_gradients);
}

FCLayer::~FCLayer()
{
    destroyDescriptors();
    hipblasDestroy(cublas_handle);
}