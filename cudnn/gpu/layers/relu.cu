#include "hip/hip_runtime.h"
#include "relu.h"
#include <cstdio>

__global__
void relu_forward_gpu(float *inp, float *out, int sz_out){
    int ind = blockDim.x*blockIdx.x + threadIdx.x;

    if (ind < sz_out){
        out[ind] = fmaxf(0, inp[ind]);
    }
}

__global__
void relu_backward_gpu(float* gradient_out, float* gradient_in, float* forward_input, int sz_out){
    int ind = blockDim.x*blockIdx.x + threadIdx.x;

    if (ind < sz_out){
        gradient_out[ind] = (forward_input[ind] > 0) ? gradient_in[ind] : 0;
    }
}

ReLU::ReLU(int _sz_out){
    sz_out = _sz_out;

    n_blocks = (sz_out + block_size - 1) / block_size;
}

void ReLU::forward(float* _inp, float* _out){
    forward_input = _inp;
    relu_forward_gpu<<<n_blocks, block_size>>>(_inp, _out, sz_out);
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error in ReLU forward: %s\n", hipGetErrorString(err));
        exit(1);
    }
}

void ReLU::backward(float* gradient_out, float* gradient_in) {
    relu_backward_gpu<<<n_blocks, block_size>>>(gradient_out, gradient_in, forward_input, sz_out);
    hipDeviceSynchronize();
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error in ReLU backward: %s\n", hipGetErrorString(err));
        exit(1);
    }
}
