#include <stdio.h>
#include "core/network.h"
#include <chrono>
#include "loss/loss.h"
#include "core/utils.h"

// Benchmark parameters
const int NUM_ITERATIONS = 100;
const int WARMUP_ITERATIONS = 300;

// Network parameters
const int BATCH_SIZE = 1;
const int NUM_CLASSES = 3;

const int IN_CHANNELS = 1;
const int INPUT_HEIGHT = 2;
const int INPUT_WIDTH = 3;

const int CONV_OUT_CHANNELS = 2;
const int CONV_KERNEL_SIZE = 2;
const int CONV_STRIDE = 1;
const int CONV_PADDING = 0;

// Define sizes in terms of number of elements
const int INPUT_SIZE = BATCH_SIZE * IN_CHANNELS * INPUT_WIDTH * INPUT_HEIGHT;
const int OUTPUT_SIZE = BATCH_SIZE * NUM_CLASSES;
const int INPUT_GRADIENT_SIZE = BATCH_SIZE * IN_CHANNELS * INPUT_WIDTH * INPUT_HEIGHT;


int main()
{
    std::chrono::steady_clock::time_point begin, end;

    // Initialize CUDNN
    hipdnnHandle_t cudnn;
    checkCUDNN(hipdnnCreate(&cudnn));

    // Create network
    Network model(cudnn, BATCH_SIZE, NUM_CLASSES, INPUT_WIDTH, INPUT_HEIGHT, IN_CHANNELS);

    model.addConvLayer(
        CONV_OUT_CHANNELS,
        CONV_KERNEL_SIZE,
        CONV_STRIDE,
        CONV_PADDING);

    model.addFCLayer(
        model.getFlattenedSize(),
        NUM_CLASSES);

    // Initialize cost history
    CostHistory cost_history;
    cost_history_init(&cost_history);

    // ================================
    // Input data
    // ================================
    float *input_data, *output_data;

    hipMallocManaged(&input_data, INPUT_SIZE * sizeof(float));
    hipMallocManaged(&output_data, OUTPUT_SIZE * sizeof(float));

    // Create input data with explicit layout
    for (int b = 0; b < BATCH_SIZE; b++)
    {
        for (int c = 0; c < IN_CHANNELS; c++)
        {
            for (int h = 0; h < INPUT_HEIGHT; h++)
            {
                for (int w = 0; w < INPUT_WIDTH; w++)
                {
                    int idx = ((b * IN_CHANNELS + c) * INPUT_HEIGHT + h) * INPUT_WIDTH + w;
                    input_data[idx] = (float)rand() / RAND_MAX;
                }
            }
        }
    }

    for (int i = 0; i < OUTPUT_SIZE; i++)
    {
        output_data[i] = 0.0f;
    }

    // Create dummy gradient for backward pass
    float *output_gradient = model.createDummyGradient(output_data);
    float *input_gradient;
    hipMallocManaged(&input_gradient, INPUT_GRADIENT_SIZE * sizeof(float));
    hipDeviceSynchronize();

    MSELoss loss;

    // ================================
    // Warmup run
    // ================================

    // Target data still needs size, but we get it from the model
    float *target_data;
    hipMallocManaged(&target_data, OUTPUT_SIZE * sizeof(float));

    for (int i = 0; i < OUTPUT_SIZE; i++)
    {
        target_data[i] = 1.0f;
    }

    for (int i = 0; i < WARMUP_ITERATIONS; i++)
    {
        model.zeroGradients();

        // Forward pass
        model.forward(input_data, output_data);

        // Compute loss and gradients
        float loss_value = loss.compute(output_data, target_data, OUTPUT_SIZE);

        if (i % 10 == 0)
            cost_history_add(&cost_history, loss_value);

        loss.backward(output_data, target_data, output_gradient, OUTPUT_SIZE);

        // Backward pass
        model.backwardInput(input_gradient, output_gradient);
        model.backwardParams(input_data, output_gradient);

        // Update weights
        model.updateWeights(0.001f);

        printf("Iteration %d, Loss: %f\n", i, loss_value);
    }

    plot_cost_ascii(&cost_history);

    hipDeviceSynchronize();

    // Cleanup
    hipFree(target_data);

    // ==================================
    // Forward pass
    // ==================================
    begin = std::chrono::steady_clock::now();
    for (int i = 0; i < NUM_ITERATIONS; i++)
    {
        model.forward(input_data, output_data);
    }
    hipDeviceSynchronize();
    end = std::chrono::steady_clock::now();

    double total_microseconds = std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count();
    double average_milliseconds = (total_microseconds / 1000.0) / NUM_ITERATIONS;
    double total_time = average_milliseconds;
    printf("Average forward pass time: %f ms\n", average_milliseconds);

    // ================================
    // Backward input pass
    // ================================
    begin = std::chrono::steady_clock::now();
    for (int i = 0; i < NUM_ITERATIONS; i++)
    {
        model.backwardInput(input_gradient, output_gradient);
    }
    hipDeviceSynchronize();
    end = std::chrono::steady_clock::now();

    total_microseconds = std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count();
    average_milliseconds = (total_microseconds / 1000.0) / NUM_ITERATIONS;
    total_time += average_milliseconds;
    printf("Average backward input pass time: %f ms\n", average_milliseconds);

    // ================================
    // Backward params pass
    // ================================
    begin = std::chrono::steady_clock::now();
    for (int i = 0; i < NUM_ITERATIONS; i++)
    {
        model.backwardParams(input_data, output_gradient);
    }
    hipDeviceSynchronize();
    end = std::chrono::steady_clock::now();

    total_microseconds = std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count();
    average_milliseconds = (total_microseconds / 1000.0) / NUM_ITERATIONS;
    total_time += average_milliseconds;
    printf("Average backward params pass time: %f ms\n", average_milliseconds);
    printf("Total time: %f ms\n", total_time);

    // Additional cleanup
    hipFree(input_gradient);
    hipFree(output_gradient);

    // Cleanup
    hipFree(input_data);
    hipFree(output_data);
    hipdnnDestroy(cudnn);

    return 0;
}
