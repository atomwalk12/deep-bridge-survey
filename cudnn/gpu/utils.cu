#include "hip/hip_runtime.h"
#include "utils.h"
#include <stdio.h>


bool ENABLE_DEBUG_OUTPUT = false;

void debugDescriptor(const char* name, hipdnnTensorDescriptor_t desc) {
    if (!ENABLE_DEBUG_OUTPUT) return;
    hipdnnDataType_t dtype;
    int n, c, h, w;
    int stride_n, stride_c, stride_h, stride_w;
    
    hipdnnGetTensor4dDescriptor(desc,
        &dtype,
        &n, &c, &h, &w,
        &stride_n, &stride_c, &stride_h, &stride_w);
        
    printf("%s descriptor:\n", name);
    printf("  Dimensions: %dx%dx%dx%d\n", n, c, h, w);
    printf("  Strides: %d,%d,%d,%d\n", stride_n, stride_c, stride_h, stride_w);
    fflush(stdout);
}

void debugFilterDescriptor(hipdnnFilterDescriptor_t desc) {
    if (!ENABLE_DEBUG_OUTPUT) return;
    hipdnnDataType_t dtype;
    hipdnnTensorFormat_t format;
    int k, c, h, w;
    
    hipdnnGetFilter4dDescriptor(desc,
        &dtype,
        &format,
        &k, &c, &h, &w);
        
    printf("Filter descriptor:\n");
    printf("  Dimensions: %dx%dx%dx%d\n", k, c, h, w);
    fflush(stdout);
}

void debugTensorValues(const char* label, float* device_ptr, int count) {
    if (!ENABLE_DEBUG_OUTPUT) return;
    
    float debug_values[10];  // Static size for simplicity
    int print_count = std::min(count, 10);  // Print at most 10 values
    
    hipError_t err = hipMemcpy(debug_values, device_ptr, 
                                print_count * sizeof(float), 
                                hipMemcpyDeviceToHost);
    
    if (err != hipSuccess) {
        printf("Error copying %s values: %s\n", 
               label, hipGetErrorString(err));
        return;
    }

    printf("First %d %s values: ", print_count, label);
    for(int i = 0; i < print_count; i++) {
        printf("%.4f ", debug_values[i]);
    }
    printf("\n");
    fflush(stdout);
}

void checkWeightChanges(const char* label, float* device_weights, int size) {

    static float prev_sum = 0.0f;  // Keep track of previous sum
    
    // Copy weights to host
    float* host_weights = new float[size];
    hipMemcpy(host_weights, device_weights, size * sizeof(float), hipMemcpyDeviceToHost);
    
    // Compute simple statistics
    float sum = 0.0f;
    for(int i = 0; i < size; i++) {
        sum += host_weights[i];
    }
    
    printf("%s - Weight sum: %.4f (change: %.4f)\n", 
           label, sum, sum - prev_sum);
    fflush(stdout);
    prev_sum = sum;
    
    delete[] host_weights;
}

void debugMatrixLayout(const char* label, float* device_ptr, 
                      int rows, int cols, int max_rows, int max_cols) {
    printf("\nMatrix Layout Debug for %s:\n", label);
    printf("Full dimensions: [%d × %d]\n", rows, cols);
    
    // Limit the size we're copying
    int display_rows = std::min(rows, max_rows);
    int display_cols = std::min(cols, max_cols);
    
    float* host_data = new float[rows * cols];
    hipMemcpy(host_data, device_ptr, rows * cols * sizeof(float), 
               hipMemcpyDeviceToHost);
    
    printf("\nAssuming Row-Major Layout:\n");
    for(int i = 0; i < display_rows; i++) {
        printf("Row %d: ", i);
        for(int j = 0; j < display_cols; j++) {
            printf("%8.4f ", host_data[i * cols + j]);
        }
        printf("\n");
    }
    
    printf("\nAssuming Column-Major Layout:\n");
    for(int i = 0; i < display_rows; i++) {
        printf("Row %d: ", i);
        for(int j = 0; j < display_cols; j++) {
            printf("%8.4f ", host_data[i + j * rows]);
        }
        printf("\n");
    }
    printf("\n");
    fflush(stdout);
    
    delete[] host_data;
} 


void cost_history_init(CostHistory *history) {
    history->count = 0;
    history->min = 1e9;
    history->max = -1e9;
}

void cost_history_add(CostHistory *history, float value) {
    if (history->count < MAX_HISTORY) {
        history->values[history->count++] = value;
        if (value < history->min) history->min = value;
        if (value > history->max) history->max = value;
    }
}

void plot_cost_ascii(CostHistory *history) {
    // This function was modified from:
    // https://github.com/karam-koujan/mini-pytorch/
    char graph[GRAPH_HEIGHT][GRAPH_WIDTH + 1];
    float range = history->max - history->min;
    
    // Initialize graph with spaces
    for (int i = 0; i < GRAPH_HEIGHT; i++) {
        memset(graph[i], ' ', GRAPH_WIDTH);
        graph[i][GRAPH_WIDTH] = '\0';
    }
    
    // Draw axis
    for (int i = 0; i < GRAPH_HEIGHT; i++) {
        graph[i][0] = '|';
    }
    for (int i = 0; i < GRAPH_WIDTH; i++) {
        graph[GRAPH_HEIGHT-1][i] = '-';
    }
    
    // Plot points with improved x-axis distribution
    float min_y[GRAPH_WIDTH];
    for (int i = 0; i < GRAPH_WIDTH; i++) {
        min_y[i] = history->max;
    }
    
    // First pass: find minimum values for each x position
    for (int i = 0; i < history->count; i++) {
        int x = (int)((float)i / history->count * (GRAPH_WIDTH - 2)) + 1;
        if (x < GRAPH_WIDTH && history->values[i] < min_y[x]) {
            min_y[x] = history->values[i];
        }
    }
    
    // Second pass: plot the points
    for (int x = 1; x < GRAPH_WIDTH; x++) {
        if (min_y[x] != history->max) {
            float normalized = (min_y[x] - history->min) / range;
            int y = GRAPH_HEIGHT - 2 - (int)(normalized * (GRAPH_HEIGHT - 3));
            if (y >= 0 && y < GRAPH_HEIGHT) {
                graph[y][x] = '*';
            }
        }
    }
    
    // Connect adjacent points with lines
    for (int x = 1; x < GRAPH_WIDTH - 1; x++) {
        if (min_y[x] != history->max && min_y[x+1] != history->max) {
            float norm1 = (min_y[x] - history->min) / range;
            float norm2 = (min_y[x+1] - history->min) / range;
            int y1 = GRAPH_HEIGHT - 2 - (int)(norm1 * (GRAPH_HEIGHT - 3));
            int y2 = GRAPH_HEIGHT - 2 - (int)(norm2 * (GRAPH_HEIGHT - 3));
            
            // Draw connecting line
            int start_y = (y1 < y2) ? y1 : y2;
            int end_y = (y1 < y2) ? y2 : y1;
            for (int y = start_y + 1; y < end_y; y++) {
                if (y >= 0 && y < GRAPH_HEIGHT) {
                    graph[y][x] = '|';
                }
            }
        }
    }
    
    // Print graph with axis labels
    printf("\nCost Function Over Epochs\n");
    printf("%8.4f ┐\n", history->max);
    for (int i = 0; i < GRAPH_HEIGHT; i++) {
        printf("%s\n", graph[i]);
    }
    printf("%8.4f ┴", history->min);
    for (int i = 0; i < GRAPH_WIDTH-10; i++) printf("─");
    printf(" %d epochs\n", history->count);
    
    // Print epoch markers
    printf("        ");  // Align with graph
    for (int i = 0; i <= 4; i++) {
        printf("%-12d", i * history->count / 4);
    }
    printf("\n");
    
    // Save to CSV for external plotting
    FILE *fp = fopen("cost_history.csv", "w");
    if (fp) {
        fprintf(fp, "epoch,cost\n");
        for (int i = 0; i < history->count; i++) {
            fprintf(fp, "%d,%.6f\n", i, history->values[i]);
        }
        fclose(fp);
        printf("\nCost history saved to 'cost_history.csv'\n");
    }

    fflush(stdout);
}